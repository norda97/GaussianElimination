#include "hip/hip_runtime.h"
/*****************************************************
*
* Gaussian elimination
*
* Sequential version
*
*****************************************************/
// Compile and then...
// Example run 1:   gauseq.exe -P 1 -I fast -n 16
// Example run 2:   gauseq.exe -P 0 -I rand -n 2048
// 

#include "hip/hip_runtime.h"
#include ""

#include "stdafx.h"
#include <stdio.h>
#include <stdlib.h>
#include <string.h>

#define MAX_SIZE 2048

typedef double matrix[MAX_SIZE][MAX_SIZE];

int	N;		/* matrix size		*/
int	maxnum;		/* max number of element*/
const char* Init;		/* matrix init type	*/
int	PRINT;		/* print switch		*/
matrix	A;		/* matrix A		*/
double	b[MAX_SIZE];	/* vector b             */
double	y[MAX_SIZE];	/* vector y             */

						/* forward declarations */
void work(void);
void Init_Matrix(void);
void Print_Matrix(void);
void Init_Default(void);
int Read_Options(int, char**);

hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size);

__global__ void addKernel(int *c, const int *a, const int *b)
{
    int i = threadIdx.x;
    c[i] = a[i] + b[i];
}

int main(int argc, char* argv[])
{

    // Create Cuda status
	hipError_t cudaStatus;

	int i, timestart, timeend, iter;

	Init_Default();		/* Init default values	*/
	Read_Options(argc, argv);	/* Read arguments	*/
	Init_Matrix();		/* Init the matrix	*/
	work();
	if (PRINT == 1)
		Print_Matrix();

    // hipDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }

    return 0;
}

void Init_Default() {
	N = 32;
	Init = "rand";
	maxnum = 15.0;
	PRINT = 0;
}

// Helper function for using CUDA to add vectors in parallel.
hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size)
{
    int *dev_a = 0;
    int *dev_b = 0;
    int *dev_c = 0;
    hipError_t cudaStatus;

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    // Allocate GPU buffers for three vectors (two input, one output)    .
    cudaStatus = hipMalloc((void**)&dev_c, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_a, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_b, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    // Launch a kernel on the GPU with one thread for each element.
    addKernel<<<1, size>>>(dev_c, dev_a, dev_b);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }
    
    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

Error:
    hipFree(dev_c);
    hipFree(dev_a);
    hipFree(dev_b);
    
    return cudaStatus;
}


void work(void)
{
	int i, j, k;

	/* Gaussian elimination algorithm, Algo 8.4 from Grama */
	for (k = 0; k < N; k++) { /* Outer loop */
		for (j = k + 1; j < N; j++)
			A[k][j] = A[k][j] / A[k][k]; /* Division step */
		y[k] = b[k] / A[k][k];
		A[k][k] = 1.0;
		for (i = k + 1; i < N; i++) {
			for (j = k + 1; j < N; j++)
				A[i][j] = A[i][j] - A[i][k] * A[k][j]; /* Elimination step */
			b[i] = b[i] - A[i][k] * y[k];
			A[i][k] = 0.0;
		}
	}
}

void Init_Matrix() {
	int i, j;

	printf("\nsize      = %dx%d ", N, N);
	printf("\nmaxnum    = %d \n", maxnum);
	printf("Init	  = %s \n", Init);
	printf("Initializing matrix...");

	if (strcmp(Init, "rand") == 0) {
		for (i = 0; i < N; i++) {
			for (j = 0; j < N; j++) {
				if (i == j) /* diagonal dominance */
					A[i][j] = (double)(rand() % maxnum) + 5.0;
				else
					A[i][j] = (double)(rand() % maxnum) + 1.0;
			}
		}
	}
	if (strcmp(Init, "fast") == 0) {
		for (i = 0; i < N; i++) {
			for (j = 0; j < N; j++) {
				if (i == j) /* diagonal dominance */
					A[i][j] = 5.0;
				else
					A[i][j] = 2.0;
			}
		}
	}

	/* Initialize vectors b and y */
	for (i = 0; i < N; i++) {
		b[i] = 2.0;
		y[i] = 1.0;
	}

	printf("done \n\n");
	if (PRINT == 1)
		Print_Matrix();
}

void Print_Matrix() {
	int i, j;

	printf("Matrix A:\n");
	for (i = 0; i < N; i++) {
		printf("[");
		for (j = 0; j < N; j++)
			printf(" %5.2f,", A[i][j]);
		printf("]\n");
	}
	printf("Vector b:\n[");
	for (j = 0; j < N; j++)
		printf(" %5.2f,", b[j]);
	printf("]\n");
	printf("Vector y:\n[");
	for (j = 0; j < N; j++)
		printf(" %5.2f,", y[j]);
	printf("]\n");
	printf("\n\n");
}

int Read_Options(int argc, char** argv) {
	char* prog;

	prog = *argv;
	while (++argv, --argc > 0)
		if (**argv == '-')
			switch (*++ * argv) {
			case 'n':
				--argc;
				N = atoi(*++argv);
				break;
			case 'h':
				printf("\nHELP: try sor -u \n\n");
				exit(0);
				break;
			case 'u':
				printf("\nUsage: sor [-n problemsize]\n");
				printf("           [-D] show default values \n");
				printf("           [-h] help \n");
				printf("           [-I init_type] fast/rand \n");
				printf("           [-m maxnum] max random no \n");
				printf("           [-P print_switch] 0/1 \n");
				exit(0);
				break;
			case 'D':
				printf("\nDefault:  n         = %d ", N);
				printf("\n          Init      = rand");
				printf("\n          maxnum    = 5 ");
				printf("\n          P         = 0 \n\n");
				exit(0);
				break;
			case 'I':
				--argc;
				Init = *++argv;
				break;
			case 'm':
				--argc;
				maxnum = atoi(*++argv);
				break;
			case 'P':
				--argc;
				PRINT = atoi(*++argv);
				break;
			default:
				printf("%s: ignored option: -%s\n", prog, *argv);
				printf("HELP: try %s -u \n\n", prog);
				break;
			}
}
